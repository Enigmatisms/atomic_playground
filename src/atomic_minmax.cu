#include "hip/hip_runtime.h"
#include "atomic_minmax.cuh"
#include "aligned_load_store.cuh"

namespace aop {

#define DEFINE_ATOMIC_MINMAX(Dtype, Mask, OpType, operator)                             \
__device__ Dtype atomic##OpType(Dtype* address, Dtype val) {                            \
    uint32_t* ptr32 = reinterpret_cast<uint32_t*>(address);                             \
    const uint32_t shift = (reinterpret_cast<size_t>(address) % 4) * 8;                 \
    const uint32_t mask = Mask << shift;                                                \
    uint32_t old32 = *ptr32, assumed32 = 0;                                             \
    Dtype current = 0, new_val = 0;                                                     \
    do {                                                                                \
        assumed32 = old32;                                                              \
        current = static_cast<Dtype>((old32 & mask) >> shift);                          \
        new_val = operator(current, val);                                               \
        uint32_t new32 = (old32 & ~mask) | (static_cast<uint32_t>(new_val) << shift);   \
        old32 = atomicCAS(ptr32, assumed32, new32);                                     \
    } while (assumed32 != old32);                                                       \
    return static_cast<Dtype>((old32 & mask) >> shift);                                 \
}

DEFINE_ATOMIC_MINMAX(int16_t, 0xFFFFU, Min, min)
DEFINE_ATOMIC_MINMAX(int16_t, 0xFFFFU, Max, max)
DEFINE_ATOMIC_MINMAX(uint8_t, 0xFFU, Min, min)
DEFINE_ATOMIC_MINMAX(uint8_t, 0xFFU, Max, max)

#undef DEFINE_ATOMIC_MINMAX

#define DEFINE_ATOMIC_SAFE_MINMAX(Dtype, OpType, operator)                              \
__device__ Dtype atomic##OpType##Safe(Dtype* address, Dtype val) {                      \
    uintptr_t base_addr = reinterpret_cast<uintptr_t>(address) & ~3;                    \
    uint32_t offset_bytes = reinterpret_cast<uintptr_t>(address) - base_addr;           \
    uint32_t shift = 0, mask = 0;                                                       \
    if constexpr (sizeof(Dtype) == 1) {                                                 \
        shift = offset_bytes * 8;                                                       \
        mask = 0xFFU << shift;                                                          \
    } else {                                                                            \
        shift = (offset_bytes / 2) * 16;                                                \
        mask = 0xFFFFU << shift;                                                        \
    }                                                                                   \
    Dtype current = 0;                                                                  \
    Dtype new_val = 0;                                                                  \
    uint32_t assumed32 = 0, old32 = loadAligned(base_addr, mask, shift);                \
    do {                                                                                \
        assumed32 = old32;                                                              \
        current = static_cast<Dtype>((old32 & mask) >> shift);                          \
        new_val = operator(current, val);                                               \
        uint32_t new32 = (old32 & ~mask) | (static_cast<uint32_t>(new_val) << shift);   \
        old32 = atomicCAS(reinterpret_cast<uint32_t*>(base_addr), assumed32, new32);    \
    } while (assumed32 != old32);                                                       \
    return current;                                                                     \
}

DEFINE_ATOMIC_SAFE_MINMAX(int16_t, Min, min)
DEFINE_ATOMIC_SAFE_MINMAX(int16_t, Max, max)
DEFINE_ATOMIC_SAFE_MINMAX(uint8_t, Min, min)
DEFINE_ATOMIC_SAFE_MINMAX(uint8_t, Max, max)

#undef DEFINE_ATOMIC_SAFE_MINMAX

}   // end namespace aop